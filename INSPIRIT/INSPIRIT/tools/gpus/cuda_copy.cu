
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define SRC_DEV 0
#define DST_DEV 1

#define DSIZE (8*1048576)

#define cudaCheckErrors(msg) do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) \
	{ \
		fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, hipGetErrorString(__err), __FILE__, __LINE__); \
		fprintf(stderr, "*** FAILED - ABORTING\n"); \
		exit(1); \
        } \
} while (0)

int main(int argc, char *argv[])
{
	int devcount;
	int srcdev = SRC_DEV;
	int dstdev = DST_DEV;
	int *d_s, *d_d, *h;
	int canAccessPeer = 0;
	int version;

	hipDriverGetVersion(&version);
	printf("driver version %d\n", version);
	hipRuntimeGetVersion(&version);
	printf("runtime version %d\n", version);

	hipGetDeviceCount(&devcount);
	cudaCheckErrors("cuda failure");

	if (devcount < 2)
	{
		printf("not enough cuda devices for the requested operation\n");
		return 1;
	}

	h = (int *)malloc(DSIZE*sizeof(int));
	if (h == NULL)
	{
		printf("malloc fail\n");
		return 1;
	}
	for (int i = 0; i < DSIZE; i++)
		h[i] = i;

	hipDeviceCanAccessPeer(&canAccessPeer, srcdev, dstdev);
	cudaCheckErrors("hipDeviceCanAccessPeer");

	printf("%s of %d bytes\n", canAccessPeer ? "Doing P2P transfer" : "Doing ordinary transfer", DSIZE*sizeof(int));

	hipSetDevice(srcdev);
	hipMalloc(&d_s, DSIZE*sizeof(int));
	hipMemcpy(d_s, h, DSIZE*sizeof(int), hipMemcpyHostToDevice);

	if (canAccessPeer)
		hipDeviceEnablePeerAccess(dstdev,0);
	hipSetDevice(dstdev);

	hipMalloc(&d_d, DSIZE*sizeof(int));
	cudaCheckErrors("hipMalloc fail");
	hipMemset(d_d, 0, DSIZE*sizeof(int));
	cudaCheckErrors("hipMemset fail");

	if (canAccessPeer)
		hipDeviceEnablePeerAccess(srcdev,0);

	hipMemcpyPeer(d_d, dstdev, d_s, srcdev, DSIZE*sizeof(int));
	cudaCheckErrors("hipMemcpyPeer fail");

	hipSetDevice(dstdev);
	hipMemcpy(h, d_d, DSIZE*sizeof(int), hipMemcpyDeviceToHost);
	cudaCheckErrors("hipMemcpy fail");

	for (int i = 0; i < DSIZE; i++)
		if (h[i] != i)
		{
			printf("transfer failure\n");
			return 1;
		}
	printf("transfer ok\n");
	return 0;
}
